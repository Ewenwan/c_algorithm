/* ***
 * This file realize the box filter based on shared memory but not texture memory
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

using namespace std;

#define INDX(row, col, colNum) (((row) * (colNum)) + col)

#define SIZE 100

#define BLOCKSIZE 16

#define FILTER_WIDTH 3
#define FILTER_RAD (FILTER_WIDTH / 2)
#define INNER_TILE (BLOCKSIZE - FILTER_RAD)

__global__ void boxfilter_kernel(float *dataIn, float *dataOut, int wid, int hei, const float * const __restrict__ filter)
{
	__shared__ float shMat[BLOCKSIZE][BLOCKSIZE];

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = tx + blockIdx.x * blockDim.x;
	int idy = ty + blockIdx.y * blockDim.y;

	int offset = idx + idy * blockDim.x * gridDim.x;

	if(tx < wid && ty < hei)
		shMat[ty][tx] = dataIn[offset];
	else
		shMat[ty][tx] = 0;

	/*
	// load data to shared memory
	if((tileX >= 0) && (tileX < wid) && (tileY >= 0) && (tileY < hei))
		shMat[ty][tx] = dataIn[INDX(tileY, tileX, wid)];
	else
		shMat[ty][tx] = 0;
	 */
	__syncthreads();

	float res = 0.0f;

	if((tx > FILTER_RAD) && (tx < INNER_TILE) && (ty > FILTER_RAD) && (ty < INNER_TILE))
	{
		for(int fr = -FILTER_RAD; fr < FILTER_RAD; ++fr)
		{
			for(int fc = -FILTER_RAD; fc < FILTER_RAD; ++fc)
			{
				res += shMat[fr + ty][fc + tx] * filter[INDX(fr, fc, FILTER_WIDTH)];
			}
		}

		if(idx < wid && idy < hei)
			dataOut[INDX(idy, idx, wid)] = res;
	}
}

int main(int argc, char **argv)
{
	cout << "hello world ..." << endl;

	float *img = new float [SIZE];

	for(int i = 0; i < SIZE; ++i)
		img[i] = 1;

	float *filter = new float [FILTER_WIDTH * FILTER_WIDTH];
	for(int i = 0; i < FILTER_WIDTH; ++i)
		for(int j = 0; j < FILTER_WIDTH; ++j)
		{
			filter[INDX(i, j, FILTER_WIDTH)] = 1 / (FILTER_WIDTH * FILTER_WIDTH);
		}



	int width = 10;
	int height = 10;

	float *d_in, *d_out;
	hipError_t cudaState = hipSuccess;
	cudaState = hipMalloc((void **)&d_in, sizeof(float) * width * height);
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&d_out, sizeof(float) * width * height);
	assert(cudaState == hipSuccess);

	// copy data from host to device
	cudaState = hipMemcpy(d_in, img, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	float *d_filter ;
	cudaState = hipMalloc((void **)&d_filter, FILTER_WIDTH * FILTER_WIDTH * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(d_filter, filter, FILTER_WIDTH * FILTER_WIDTH * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);

	dim3 threadPerBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 blockPerGrid;
	blockPerGrid.x = (width + threadPerBlock.x - 1) / BLOCKSIZE;
	blockPerGrid.y = (height + BLOCKSIZE - 1) / BLOCKSIZE;

	boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(d_in, d_out, width, height, d_filter);

	// copy data back to host
	cudaState = hipMemcpy(img, d_out, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);

	for(int i = 0; i < 10; ++i)
	{
		for(int j = 0; j < 10; ++j)
			cout << img[INDX(i, j, 10)] << "; ";
		cout << endl;
	}

	delete [] img;

	return 0;
}
