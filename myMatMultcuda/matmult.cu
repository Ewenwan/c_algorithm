#include "hip/hip_runtime.h"
#include "matmult.h"

__global__ void matKernel(float *L, float *R, float *C, int m, int k, int n)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//int offset = x + y * blockDim.x * gridDim.x;
	int offset = x + y * n;

	int a = x;    // horizontal
	int b = y;    // vertical

	float sum = 0;

	while(a < n)   // calculate all blocks along the x direction
	{
		while(b < m)    // calculate all blocks along y direction
		{
			sum = 0;
			for(int i = 0; i < k; ++i)
			{
				sum += (L[b * k + i] * R[a + i * n]);
			}
			C[offset] = sum;
			b += blockDim.y * gridDim.y;
			offset = a + b * n;
		}
//		__syncthreads();
		a += blockDim.x * gridDim.x;
		b = y;
		offset = a + b * n;
	}
//	__syncthreads();
}

void matMult::matMultF(matMult *L, matMult *R, matMult *C)
{
	hipError_t cudaState = hipSuccess;

	const int SIZEL = L->col * L->row;
	const int SIZER = R->col * R->row;
	const int SIZEC = C->row * C->col;
	// prepare for data, malloc memory on GPU
	cudaState = hipMalloc((void **)&(L->A_d), SIZEL * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&(R->A_d), SIZER * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&(C->A_d), SIZEC * sizeof(float));
	assert(cudaState == hipSuccess);

	// copy data from host to device
	cudaState = hipMemcpy(L->A_d, L->A, SIZEL * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(R->A_d, R->A, SIZER * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMemset(C->A_d, 0, SIZEC * sizeof(float));
	assert(cudaState == hipSuccess);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start time record
	hipEventRecord(start, 0);

	dim3 blockPerGrid(4, 4);
	dim3 threadPerBlock(16, 16);
	//const int Width = blockPerGrid.x * threadPerBlock.x;

	// launch the kernel
	matKernel<<<blockPerGrid, threadPerBlock>>>(L->A_d, R->A_d, C->A_d, L->row, L->col, R->col);

	// stop time record
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "GPU tiem : " << elapsedTime << " ms" << endl;

	// copy result from device back to host
	cudaState = hipMemcpy(C->A, C->A_d, SIZEC * sizeof(float), hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);


	hipFree(L->A_d);
	hipFree(R->A_d);
	hipFree(C->A_d);
}
