#include "hip/hip_runtime.h"
#include "GFilter.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// declare texture memory reference
texture<float, hipTextureType2D, hipReadModeElementType> texA, texB;

__constant__ float d_eps;

// declare constant memory

#define BLOCK_SIZE 32
// TODO : change this macro define to kernel parameter to decide the size of shared memory in boxfilter_kernel
#define BOXWIDTH 17
// R : row, C : col, N : colNums
#define INDX(R, C, N) ((R) * (N) + (C))

/*
#define FILTER 0
#define FILTER_WIDTH 5
#define FILTER_RAD (FILTER_WIDTH / 2)

#define BLOCKSIZE 32
#define INNER_TILE_WIDTH (BLOCKSIZE - FILTER_RAD * 2)

#define INDX(row, col, wid) (((row) * (wid)) + (col))

__global__ void boxfilter_kernel(const float * const inputChannel, float *const outputChannel, int wid, int hei, const float * __restrict__ const filter)
{
    // shared memory data (+1 to avoid bank conflict
    __shared__ float sh_matrix[BLOCKSIZE+1][BLOCKSIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tileX = bx * INNER_TILE_WIDTH + tx;   // cols
    int tileY = by * INNER_TILE_WIDTH + ty;   // rows

    // if tileX_image < 0, then it must be outside of the image
    int tileX_image = tileX - FILTER_RAD;
    int tileY_image = tileY - FILTER_RAD;

    // fill shared memory
    if((tileX_image >= 0) && (tileX_image < wid) && (tileY_image >= 0) && (tileY_image < hei))
        sh_matrix[ty][tx] = inputChannel[INDX(tileY_image, tileX_image, wid)];
    else
        sh_matrix[ty][tx] = 0;

    __syncthreads();

    // Do convolution (only in the inner tile of block)
    float res = 0.0f;
    if(tx < INNER_TILE_WIDTH && ty < INNER_TILE_WIDTH)
    {
        for(int fr = 0; fr < FILTER_WIDTH; ++fr)
            for(int fc = 0; fc < FILTER_WIDTH; ++fc)
                res += sh_matrix[fr + ty][fc + tx] * filter[INDX(fr, fc, FILTER_WIDTH)];

        if(tileX < wid && tileY < hei)
            outputChannel[INDX(tileY_image, tileX_image, wid)] = res;
    }
}
*/
// calculate the variance of varI based on texture memory
// out : corrIp and convIp,
__global__ void variance_kernel(float *out, float *lhs, int wid, int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx >= wid || idy >= hei)
        return ;

    float val = 0.f;

    val = tex2D(texA, idy, idx);
    val *= val;

    val = lhs[INDX(idy, idx, wid)] - val;

    out[INDX(idy, idx, wid)] = val;
}

__global__ void covariance_kernel(float *out, float *lhs, int wid, int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx >= wid || idy >= hei)
        return ;

    float val = 0.0f;

    val = tex2D(texA, idy, idx);
    val *= tex2D(texB, idy, idx);

    out[INDX(idy, idx, wid)] = lhs[INDX(idy, idx, wid)] - val;
}

// input 3 variables : meanP, meanI and "a", the former two are stored in texture now
__global__ void calcuB_kernel(float *out, float *in, int wid, int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx >= wid || idy >= hei)
        return ;

    float val;
    val = in[INDX(idy, idx, wid)];

    val *= tex2D(texA, idy, idx);

    val = tex2D(texB, idy, idx) - val;

    out[INDX(idy, idx, wid)] = val;
}

__device__ int clamp(int a, int b, int c)
{
    int temp = a > b ? a : b;
    return (temp < c ? temp : c);
}

// boxfilter based on share memory
__global__ void boxfilter_kernel(float * const d_in, float * const d_out, int wid, int hei, const float * __restrict__ filter, const int filterWidth)
{
    const unsigned int x0 = threadIdx.x;
    const unsigned int y0 = threadIdx.y;

    const int tx = x0 + blockIdx.x * blockDim.x;
    const int ty = y0 + blockIdx.y * blockDim.y;

    if(tx >= wid || ty >= hei)
        return ;

    const int filterRadius = filterWidth / 2;
    float value = 0.f;

    __shared__ float shMat[BLOCK_SIZE + BOXWIDTH - 1][BLOCK_SIZE + BOXWIDTH - 1];

    int fx, fy;
    // case 1 : upper left
    fx = tx - filterRadius;    // col
    fy = ty - filterRadius;    // row
    fx = clamp(fx, 0, wid);
    fy = clamp(fy, 0, hei);
    shMat[x0][y0] = d_in[INDX(fy, fx, wid)];

    // case 2 : upper right
    fx = tx + filterRadius;
    fy = ty - filterRadius;
    fx = clamp(fx, 0, wid);
    fy = clamp(fy, 0, hei);
    shMat[x0 + filterWidth - 1][y0] = d_in[INDX(fy, fx, wid)];

    // case 3 : lower left
    fx = tx - filterRadius;
    fy = ty + filterRadius;
    fx = clamp(fx, 0, wid);
    fy = clamp(fy, 0, hei);
    shMat[x0][y0 + filterWidth - 1] = d_in[INDX(fy, fx, wid)];

    // case 4 : lower right
    fx = fx + filterRadius;
    fy = fy + filterRadius;
    fx = clamp(fx, 0, wid);
    fy = clamp(fy, 0, hei);
    shMat[x0+filterWidth - 1][y0 + filterWidth - 1] = d_in[INDX(fy, fx, wid)];

    __syncthreads();

    for(int i = 0; i < filterWidth; ++i)
        for (int j = 0; j < filterWidth; ++j)
        {
            value += filter[INDX(j, i, filterWidth)] * shMat[x0+i][y0+j];
        }
    d_out[INDX(ty, tx, wid)] = value;
}

// calculate the image I based on texture memory
__global__ void boxfilterTI_kernel(float *out, int wid, int hei, const size_t pitch, const int fWid, const int fHei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    float outputVal = 0.0f;

    if(idx < wid && idy < hei)
    {
        for(int i = -fWid; i <= fWid; ++i)
        {
            for(int j = -fHei; j <= fHei; ++j)
                outputVal += tex2D(texA, idy + j, idx + i);
        }
        outputVal /= ((2 * fWid + 1) * (2 * fHei + 1));

        int offset = idy * pitch / sizeof(float) + idx;
        out[offset] = outputVal;
    }
}
__global__ void boxfilterTP_kernel(float *out, int wid, int hei, const size_t pitch, const int fWid, const int fHei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    float outputVal = 0.0f;

    if(idx < wid && idy < hei)
    {
        for(int i = -fWid; i <= fWid; ++i)
        {
            for(int j = -fHei; j <= fHei; ++j)
                outputVal += tex2D(texB, idy + j, idx + i);
        }
        outputVal /= ((2 * fWid + 1) * (2 * fHei + 1));

        int offset = idy * pitch / sizeof(float) + idx;
        out[offset] = outputVal;
    }
}

__global__ void elemwisemult_kernel(float *out, size_t pitch, const int wid, const int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    int offset = idy * pitch / sizeof(float) + idx;

    float res = 0.f;

    if(idx < wid && idy < hei)
    {
        res = tex2D(texA, idy, idx) * tex2D(texB, idy, idx);
    }

    out[offset] = res;
}

/*
__global__ void elemwisemult_kernel(float *out, float *inA, float *inB, size_t pitch, const int wid, const int hei)
{
    extern __shared__ float A[];
    extern __shared__ float B[];

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    //int index = idy * blockDim.x * gridDim.x + idx;
    int offset = idy * pitch / sizeof(float) + idx;

    int sharedIdx = threadIdx.y * blockDim.x + threadIdx.x;

    if(idx < wid && idy < hei)
    {
        A[sharedIdx] = inA[offset];
        B[sharedIdx] = inB[offset];
        //A[threadIdx.x][threadIdx.y] = inA[index];
        //B[threadIdx.x][threadIdx.y] = inB[index];

        __syncthreads();

        out[offset] = A[sharedIdx] * B[sharedIdx];
    }
}
*/

__global__ void elemwisemultSame_kernel(float *out, const size_t pitch, const int wid, const int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    float outputVal = 0.0f;

    if(idx < wid && idy < hei)
    {
        outputVal = tex2D(texA, idy, idx);
        outputVal *= outputVal;
    }

    int offset = idy * pitch / sizeof(float) + idx;
    out[offset] = outputVal;
}

/*
// dir means the operation : 0 for add, 1 for subtraction
__global__ void elewiseAdd_kernel(float *lhs, float *rhs, int dir, int wid, int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    int index = idy * wid + idx;

    if(idx < wid && idy < hei)
    {
        if(dir == 0)   // for operation +
            lhs[index] += rhs[index];
        else if(dir == 1)    // for operation -
            lhs[index] -= rhs[index];
        else
            lhs[index] = lhs[index];
    }
}
*/

// the img I is binded to texA, so the source operators are in1, in2 only
// in1 : meanA, in2 : meanB
__global__ void finnalCal_kernel(float * const out, const float * const in1, const float * const in2, int wid, int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx >= wid || idy >= hei)
        return ;

    float val = 0.f;

    val = in1[INDX(idy, idx, wid)] * tex2D(texA, idy, idx);

    val += in2[INDX(idy, idx, wid)];

    out[INDX(idy, idx, wid)] = val;
}

__global__ void elewiseDiv_kernel(float *lhs, float *rhs, int wid, int hei)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    //int index = idy * wid + idx;

    if(idx < wid && idy < hei)
    {
        lhs[INDX(idy, idx, wid)] /= rhs[INDX(idy, idx, wid)] + d_eps;
    }
}

// allocate page-locked memory on host hipHostAlloc, use hipHostFree to free them
void GFilter::prepareMem(float *dataInI, float *dataInP)
{

}

void GFilter::createfilter()
{
    //const int filterSize = (2 * rad + 1) * (2 * rad + 1);
    filter = new float [FILTERSIZE];
    for(int i = 0; i < FILTERSIZE; ++i)
        filter[i] = 1.0 / FILTERSIZE;
}

/*
void GFilter::boxfilter(float *d_in, float *d_out, int rad, const float * const __restrict__ filter)
{
    dim3 threadPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 blockPerGrid((width + threadPerBlock.x - 1) / BLOCKSIZE, (height + threadPerBlock.y - 1) / BLOCKSIZE);
}
*/

void GFilter::guidedfilter()
{
    // copy data from host to device
    // allocate pitch memory on device use hipMallocPitch
    size_t pitch = 0;
    hipError_t cudaState = hipSuccess;
    cudaState = hipMallocPitch((void **)&d_imgI, &pitch, width, height);   // pitch' unit is bytes !
    assert(cudaState == hipSuccess);
    cudaState = hipMallocPitch((void **)&d_imgP, &pitch, width, height);
    assert(cudaState == hipSuccess);
    // allocate temp data on device use hipMalloc
    cudaState = hipMalloc((void **)&d_meanI, sizeof(float) * width * height);
    assert(cudaState == hipSuccess);
    cudaState = hipMalloc((void **)&d_meanP, sizeof(float) * width * height);
    assert(cudaState == hipSuccess);
    cudaState = hipMalloc((void **)&d_tempA, sizeof(float) * width * height);
    assert(cudaState == hipSuccess);
    cudaState = hipMalloc((void **)&d_tempB, sizeof(float) * width * height);
    assert(cudaState == hipSuccess);
    // copy data from host to device use hipMemcpy2D
    // the dst pitch and src pitch all are in bytes
    cudaState = hipMemcpy2D(d_imgI, pitch, h_imgI, sizeof(float) * width, sizeof(float) * width, height, hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);
    cudaState = hipMemcpy2D(d_imgP, pitch, h_imgP, sizeof(float) * width, sizeof(float) * width, height, hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);

    // prepare the texture memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    texA.addressMode[0] = texA.addressMode[1] = hipAddressModeBorder;
    // bind the 2D Pitch memory to 2D texture memory
    cudaState = hipBindTexture2D(NULL, texA, d_imgI, channelDesc, width, height, pitch);
    assert(cudaState == hipSuccess);
    cudaState = hipBindTexture2D(NULL, texB, d_imgP, channelDesc, width, height, pitch);
    assert(cudaState == hipSuccess);

    /*
    // prepare the filter on device constant memory by using hipMemcpyToSymbol
    createfilter();
    cudaState = hipMemcpyToSymbol(d_);
    */
    createfilter();
    cudaState = hipMalloc((void **)&d_filter, FILTERSIZE * sizeof(float));
    assert(cudaState == hipSuccess);
    cudaState = hipMemcpy(d_filter, filter, FILTERSIZE * sizeof(float), hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);


    // prepare the kernel launching parameters
    dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blockPerGrid;
    blockPerGrid.x = (width + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (height + threadPerBlock.y - 1) / threadPerBlock.y;

    // launch boxfilter kernel with stream technology
    hipStream_t st[4];
    // create stream use hipStreamCreate
    for(int i = 0; i < 4; ++i)
    {
        cudaState = hipStreamCreate(&st[i]);
        assert(cudaState == hipSuccess);
    }

    boxfilterTI_kernel<<<blockPerGrid, threadPerBlock, 0, st[0]>>>(d_meanI, width, height, pitch, rad, rad);
    boxfilterTP_kernel<<<blockPerGrid, threadPerBlock, 0, st[1]>>>(d_meanP, width, height, pitch, rad, rad);
    elemwisemultSame_kernel<<<blockPerGrid, threadPerBlock, 0, st[2]>>>(d_tempA, pitch, width, height);
    elemwisemult_kernel<<<blockPerGrid, threadPerBlock, 0, st[3]>>>(d_tempB, pitch, width, height);

    // synchronize all above  streams use cudaDeviceSyncthronize()
    cudaState = hipDeviceSynchronize();
    assert(cudaState == hipSuccess);

    // calculate the corrI and corrIp
    boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempA, d_tempA, width, height, d_filter, FILTERWIDTH);
    boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempB, d_tempB, width, height, d_filter, FILTERWIDTH);

    // change the texA to d_meanI
    // Unbind the texA from origin d_imgI
    cudaState = hipUnbindTexture(texA);
    assert(cudaState == hipSuccess);
    // rebind the texA to d_meanI
    // the pitch of d_meanI is width * sizeof(float)
    cudaState = hipBindTexture2D(NULL, texA, d_meanI, channelDesc, width, height, width * sizeof(float));
    assert(cudaState == hipSuccess);

    // calculation the variance of I: varI
    variance_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempA, d_tempA, width, height);

    // rebind the texB to d_meanP
    // unbind the texB from d_imgP;
    cudaState = hipUnbindTexture(texB);
    assert(cudaState == hipSuccess);
    // bind the texB to d_meanp
    cudaState = hipBindTexture2D(NULL, texB, d_meanP, channelDesc, width, height, sizeof(float) * width);
    assert(cudaState == hipSuccess);
    // calculate the covariance of I and P based on global memory
    covariance_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempB, d_tempB, width, height);

    // calculate the a & b
    // calculate the "a" and store it in d_tempB
    // copy eps to device constant memory which is d_eps
    cudaState = hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(float), NULL, hipMemcpyDefault);
    assert(cudaState == hipSuccess);
    // launch the kernel function
    elewiseDiv_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempB, d_tempA, width, height);

    /*
    // rebind texB to above "a"
    cudaState = hipUnbindTexture(texB);
    assert(cudaState == hipSuccess);
    // bind the texB to "a" which is in d_tempB
    cudaState = hipBindTexture2D(NULL, texB, d_tempB, channelDesc, sizeof(float) * width);
    assert(cudaState == hipSuccess);
    */

    // calculate the "b" and store it in tempA
    // launch the the kernel function
    calcuB_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempA, d_tempB, width, height);


    // calculate the boxfilter of "a"  &  "b"
    // calculate meanA based on d_tempB
    boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempB, d_tempB, width, height, d_filter, FILTERWIDTH);
    // calculate meaB based on d_tempA
    boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempA, d_tempA, width, height, d_filter, FILTERWIDTH);

    // calculate the finnal output: guided filtering result
    Res = new float [width * height];
    // rebind the texA to d_imgI
    // unbind the texA from d_meanI
    cudaState = hipUnbindTexture(texA);
    assert(cudaState == hipSuccess);
    // rebind the texA to d_imgI
    cudaState = hipBindTexture2D(NULL, texA, d_imgI, width, height, pitch);
    assert(cudaState == hipSuccess);
    finnalCal_kernel<<<blockPerGrid, threadPerBlock>>>(d_meanI, d_tempB, d_tempA, width, height);

    // Finnal step : copy data from device back to host
    cudaState = hipMemcpy(Res, d_meanI, sizeof(float) * width * height, hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);


    // Unbind all textures
    cudaState = hipUnbindTexture(texB);
    assert(cudaState == hipSuccess);
    cudaState = hipUnbindTexture(texA);
    assert(cudaState == hipSuccess);


    // destory streams use cudaStreamDestory
    for(int i = 0; i < 4; i++)
    {
        cudaState = hipStreamDestroy(st[i]);
        assert(cudaState == hipSuccess);
    }

    if(d_imgI)
        hipFree(d_imgI);
    if(d_imgP)
        hipFree(d_imgP);
    if(d_tempA)
        hipFree(d_tempA);
    if(d_tempB)
        hipFree(d_tempB);
    if(d_meanI)
        hipFree(d_meanI);
    if(d_meanP)
        hipFree(d_meanP);
    if(d_filter)
        hipFree(d_filter);

}

GFilter::~GFilter()
{
    /*
    if(d_imgI)
        hipFree(d_imgI);
    if(d_imgP)
        hipFree(d_imgP);
    if(d_tempA)
        hipFree(d_tempA);
    if(d_tempB)
        hipFree(d_tempB);
    if(d_meanI)
        hipFree(d_meanI);
    if(d_meanP)
        hipFree(d_meanP);
    if(d_filter)
        hipFree(d_filter);

    if(filter)
        delete [] filter;

    if(Res)
        delete [] Res;
    */
}
