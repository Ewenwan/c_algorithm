
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <hipblas.h>

using namespace std;

__constant__ float SUM_C_d[10 * 10];

int main(void)
{
	hipblasHandle_t cublasH;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipError_t cudaState = hipSuccess;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	cublas_status = hipblasCreate(&cublasH);
	assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

	float *L = new float [512 * 512];
	float *R = new float [512 * 512];
	float *SUM = new float [512 * 512]();
	// initialize the R and L
	for(int i = 0; i < 512 * 512; i++)
	{
		L[i] = 2;
		R[i] = 1;
	}
	int row = 512;
	int rank = 512;
	int col = 512;

	const int SIZEL = 512 * 512;
	const int SIZER = 512 * 512;
	const int SIZES = 512 * 512;
	// malloc memory on Device
	float *L_d, *R_d, *SUM_d;
	cudaState = hipMalloc((void **)&L_d, SIZEL * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&R_d, SIZER * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&SUM_d, SIZES * sizeof(float));
	assert(cudaState == hipSuccess);

	// copy data from host to device
	cudaState = hipMemcpy(L_d, L, SIZEL * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(R_d, R, SIZER * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMemset(SUM_d, 0, SIZES * sizeof(float));
	assert(cudaState == hipSuccess);

	float alpha = 1.0, beta = 1.0;

	// Test contant memory
	// copy data from host to constant memory


	// start timing
	hipEventRecord(start, 0);


	// Call Blas APIs : cublasSgemm()
	cublas_status = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, row, col, rank, &alpha, L_d, row, R_d, rank, &beta, SUM_d, row);
	assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

	// stop timing
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "GPU time : " << elapsedTime << " ms" << endl;

	// copy data from device back to host
	cudaState = hipMemcpy(SUM, SUM_d, SIZES * sizeof(float), hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);

	int count = 0;

	for(int i = 0; i < 512 * 512; ++i)
	{
		if(SUM[i] != 0)
			++count;
		if(i < 10)
			cout << SUM[i] << endl;
	}

	cout << "count = " << count << endl;

	cublas_status = hipblasDestroy(cublasH);
	return 0;
}
