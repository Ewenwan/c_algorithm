#include "hip/hip_runtime.h"
/* ************
 *  This file is attempted to test the relation ship between the bounded device memory and bounding texture memory.
 *  Result :
 *  When the values in device memory changed, the texture memory should change as well, as long as the BOUNDING relationship is working.
 *	When you bind linear memory to texture, u should use tex1Dfetch to fetch the contents in texture memory.
 * 	tex1D API is used only to one-dimension array
 */



#include <iostream>
#include <cassert>

using namespace std;

#define SIZE 10

texture<float, hipTextureType1D, hipReadModeElementType> texIn;

__global__ void testTex(float *Out)
{
	int idx = threadIdx.x;

	if(idx < SIZE)
		Out[idx] = tex1Dfetch(texIn, idx);
}

void printMat(float *data)
{
	for(int i = 0; i < SIZE; i++)
		cout << data[i] << "; ";
	cout << endl;
}

int main(void)
{
	float *data = new float [SIZE];
	float *out = new float [SIZE];

	// first version
	for(int i = 0; i < SIZE; i++)
		data[i] = i;

	cout << "Input : " << endl;
	printMat(data);

	hipError_t cudaState = hipSuccess;

	// malloc global memory on GPU
	float *d_data, *d_out;
	cudaState = hipMalloc((void **)&d_data, sizeof(float) * SIZE);
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(d_data, data, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&d_out, sizeof(float) * SIZE);
	assert(cudaState == hipSuccess);

	// create texture memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	texIn.addressMode[0] = hipAddressModeWrap;
	cudaState = hipBindTexture(NULL, texIn, d_data, channelDesc, SIZE * sizeof(float));
	assert(cudaState == hipSuccess);

	// launch kernel function
	testTex<<<1, SIZE>>>(d_out);

	// copy data back to host
	cudaState = hipMemcpy(out, d_out, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);

	// print the result
	printMat(out);

	/*
	// change the input data
	cudaState = hipMemset(d_data, 0, SIZE * sizeof(float));   // Caution : every byte is set to 1, the unit is BYTE !
	assert(cudaState == hipSuccess);
*/

	for(int i = 0; i < SIZE; i++)
		data[i] = 2 * i;

	cudaState = hipMemcpy(d_data, data, SIZE * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);

	testTex<<<1, SIZE>>>(d_out);

	// copy data back to host
	cudaState = hipMemcpy(out, d_out, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);

	// print the result
	printMat(out);


	hipUnbindTexture(texIn);

	delete [] data;
	delete [] out;

	return 0;
}
