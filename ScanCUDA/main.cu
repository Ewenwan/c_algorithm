
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

/* ******************
 * Author : smher
 * Data   : 2017. 08. 01
 * Description :
 * 		This file implement the all-prefix-sums (scan) algorithm parallel. based on "GPU Gem 3" chapter 39.
 *
 * 		The Version 2 ( Up Sweep and Down Sweep cannot be fit to not power of 2 lenght array !!!
 *
 * 		TODO:
 * 		    There is a lot of room for optimization as for Versioin 2 !!!
 */

#define SIZE 128
//#define SIZE 127   // Not fit for Version 2

using namespace std;

//__device__ void UpSweep(float *data, int width)
__global__ void UpSweep(float *data, int width)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int index = idy * (blockDim.x * gridDim.x) + idx;

	float times = roundf(log2f(width));

	for(int i = 0; i < times; i++)
	{
		int offsetA = powf(2, i+1);
		int offsetB = powf(2, i);
		//if(index < width-offsetA)
		if(index < width)
		{
			if(index % offsetA == 0)
				data[index + offsetA - 1] += data[index + offsetB - 1];
		}

		__syncthreads();
	}
}

__global__ void DownSweep(float *data, int width)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int index = idy * (blockDim.x * gridDim.x) + idx;

	float times = roundf(log2f(width));
    //assert(times == 7);
	//float temp = 0;

	if(index == (width - 1))
		data[index] = 0;

	for(int i = times - 1; i >= 0; i--)
	{
		int offsetA = powf(2, i + 1);
		int offsetB = powf(2, i);

		if(index <= (width - offsetA))
		{
			if(index % offsetA == 0)
			{
				float temp = data[index + offsetB - 1];
				data[index + offsetB - 1] = data[index + offsetA - 1];
				data[index + offsetA - 1] += temp;
			}
		}
		__syncthreads();
	}
}

void scan(float *dataIn, int width)
{
    dim3 threadPerBlock(8, 8);
    dim3 blockPerGrid(1, 2);
    UpSweep<<<blockPerGrid, threadPerBlock>>>(dataIn, SIZE);
    DownSweep<<<blockPerGrid, threadPerBlock>>>(dataIn, SIZE);
}

__global__ void scanShared(float *g_odata, float *g_idata, int n)
{
	 extern __shared__ float temp[]; // allocated on invocation
	 int thid = threadIdx.x;
	 int pout = 0, pin = 1;
	 // Load input into shared memory.
	 // This is exclusive scan, so shift right by one
	 // and set first element to 0
	 temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
	 __syncthreads();
	 for (int offset = 1; offset < n; offset *= 2)
	 {
		   pout = 1 - pout; // swap double buffer indices
		   pin = 1 - pout;
		   if (thid >= offset)
			 temp[pout*n+thid] += temp[pin*n+thid - offset];
		   else
			 temp[pout*n+thid] = temp[pin*n+thid];
		   __syncthreads();
	 }
	 g_odata[thid] = temp[pout*n+thid]; // write output
}


__global__ void scanFirst(float *data, int width)
{
	float times = roundf(log2f(width));

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int index = idy * (blockDim.x * gridDim.x) + idx;

	//assert(times == 7);    // for sure that the time is correct

	for(int i = 0; i < times; i++)
	{
		//int k = 2 << i;  // the powf(2, i)
		int k = powf(2, i);    // k is the stride

		if(index >= k && index < width)
		{
			data[index] = data[index - k] + data[index];
		}
		__syncthreads();
	}
}

/*
 * TODO
__global__ void scanMat(float *dataMat, int row, int col)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;
	int index = idy * (blockDim.x * gridDim.x) + idx;
}
*/

void print(float *src, int width)
{
	for(int i = 0; i < width; i++)
		cout << src[i] << "  ";
	cout << endl;
}

void scanSeq(float *src, float *dst, int width)
{
	for(int i = 0; i < width; i++)
		dst[i] = dst[i-1] + src[i];
}

int main(int argc, char **argv)
{
	cout << "hello world ..." << endl;
	float *src = new float [SIZE];
	float *dst = new float [SIZE]();
	for(int i = 0; i < SIZE; i++)
		src[i] = i;

	// prepare the cuda event
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float elapsedTime;

	// prepare datas on device
	float *srcD;
	float *dstD;
	hipError_t cudaState = hipSuccess;
	cudaState = hipMalloc((void **)&srcD, SIZE * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(srcD, src, SIZE * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&dstD, SIZE * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMemset(dstD, 0, SIZE * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(dstD, srcD, SIZE * sizeof(float), hipMemcpyDeviceToDevice);

	// sequential version
    cout << "Sequential Versioin : " << endl;
	scanSeq(src, dst, SIZE);

	print(dst, SIZE);

    cout << "Version 1 " << endl;
	hipEventRecord(start, 0);
	// The first parallel version of scan algorithm
	dim3 threadPerBlock(8, 8);
	dim3 blockPerGrid(1,2);
	scanFirst<<<blockPerGrid, threadPerBlock>>>(dstD, SIZE);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime, start, end);

	cudaState = hipMemcpy(dst, dstD, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);
	print(dst, SIZE);

	cout << "Version 1 Parallel Scan: " << elapsedTime << " ms" << endl;

    // Test for complete scan function
    cout << "Version 2: " << endl;
    cudaState = hipMemcpy(dstD, src, SIZE * sizeof(float), hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);
    hipEventRecord(start, 0);

    scan(dstD, SIZE);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsedTime, start, end);
    cudaState = hipMemcpy(dst, dstD, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    assert(cudaState == hipSuccess);
    cout << "Exclusive Version : " << endl;
    print(dst, SIZE);

    // change the exclusive to inclusive version Scan
    for(int i = 0; i < SIZE - 1; i++)
        dst[i] = dst[i + 1];
    dst[SIZE - 1] += src[SIZE - 1];
    cout << "Inclusive Version : " << endl;
    print(dst, SIZE);

    cout << "Version 2 Parallel Scan: " << elapsedTime << " ms" << endl;

    // Debug for UpSweep and DownSweep Kernels
    /*
	// Test for UpSweep Functioin
	cudaState = cudaMemcpy(dstD, src, SIZE * sizeof(float), cudaMemcpyHostToDevice);
	assert(cudaState == cudaSuccess);
	UpSweep<<<blockPerGrid, threadPerBlock>>>(dstD, SIZE);
	cudaState = cudaMemcpy(dst, dstD, SIZE * sizeof(float), cudaMemcpyDeviceToHost);
	assert(cudaState == cudaSuccess);
	print(dst, SIZE);

	// Test for DownSweep Functioni
	DownSweep<<<blockPerGrid, threadPerBlock>>>(dstD, SIZE);
	cudaState = cudaMemcpy(dst, dstD, SIZE * sizeof(float), cudaMemcpyDeviceToHost);
	assert(cudaState == cudaSuccess);
    */

	/*
	// test for the first shared memory version
	//scanShared<<<blockPerGrid, threadPerBlock>>>(dstD, srcD, SIZE);     // Error
	//scanShared<<<1, SIZE>>>(dstD, srcD, SIZE);
	cudaState = cudaMemcpy(dst, dstD, SIZE * sizeof(float), cudaMemcpyDeviceToHost);
	assert(cudaState == cudaSuccess);
	print(dst, SIZE);
	*/

	// free the resources
	delete [] src;
	delete [] dst;

	hipFree(srcD);
	hipFree(dstD);

	hipEventDestroy(start);
	hipEventDestroy(end);

	return 0;
}
