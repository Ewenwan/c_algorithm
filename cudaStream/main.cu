#include "hip/hip_runtime.h"
#include <iostream>

#define N (256 * 256)
#define FULL_DATA_SIZE (N*20)

using namespace std;

__global__ void kernel(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N)
	{
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(int argc, char **argv)
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	cout << "which Device = " << whichDevice << endl;
	hipGetDeviceProperties(&prop, whichDevice);
	if(!prop.deviceOverlap)
	{
		cout << "Device will not handle overlaps" << endl;
		return 0;
	}

	// create cuda event to calculate the time
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// initialize the streams
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	// malloc data
	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	hipMalloc((void **)&dev_a0, N * sizeof(int));
	hipMalloc((void **)&dev_b0, N * sizeof(int));
	hipMalloc((void **)&dev_c0, N * sizeof(int));
	hipMalloc((void **)&dev_a1, N * sizeof(int));
	hipMalloc((void **)&dev_b1, N * sizeof(int));
	hipMalloc((void **)&dev_c1, N * sizeof(int));

	// allocate page-locked memory , used to stream
	hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for(int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for(int i = 0; i < FULL_DATA_SIZE; i+=N*2)
	{
		hipMemcpyAsync(dev_a1,host_a+i + N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b1,host_b+i + N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_a0,host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b0,host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);

		kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);

		hipMemcpyAsync(host_c+i + N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "total time : " << elapsedTime << " ms" << endl;

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);

	// destroy stream
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	return 0;
}

/*  **** with no gpu scheduling **** */
/*
int main(int argc, char **argv)
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	cout << "which Device = " << whichDevice << endl;
	hipGetDeviceProperties(&prop, whichDevice);
	if(!prop.deviceOverlap)
	{
		cout << "Device will not handle overlaps" << endl;
		return 0;
	}

	// create cuda event to calculate the time
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// initialize the streams
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	// malloc data
	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	hipMalloc((void **)&dev_a0, N * sizeof(int));
	hipMalloc((void **)&dev_b0, N * sizeof(int));
	hipMalloc((void **)&dev_c0, N * sizeof(int));
	hipMalloc((void **)&dev_a1, N * sizeof(int));
	hipMalloc((void **)&dev_b1, N * sizeof(int));
	hipMalloc((void **)&dev_c1, N * sizeof(int));

	// allocate page-locked memory , used to stream
	hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for(int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for(int i = 0; i < FULL_DATA_SIZE; i+=N*2)
	{
		hipMemcpyAsync(dev_a0,host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b0,host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);

		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);

		hipMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0);

		hipMemcpyAsync(dev_a1,host_a+i + N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b1,host_b+i + N, N*sizeof(int), hipMemcpyHostToDevice, stream1);

		kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

		hipMemcpyAsync(host_c+i + N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "total time : " << elapsedTime << " ms" << endl;

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);

	// destroy stream
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);



	return 0;
}
*/

/* ******* only one stream ********* */
/*
int main(int argc, char **argv)
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	cout << "which Device = " << whichDevice << endl;
	hipGetDeviceProperties(&prop, whichDevice);
	if(!prop.deviceOverlap)
	{
		cout << "Device will not handle overlaps" << endl;
		return 0;
	}

	// create cuda event to calculate the time
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// initialize the stream
	hipStream_t stream;
	hipStreamCreate(&stream);

	// malloc data
	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **)&dev_a, N * sizeof(int));
	hipMalloc((void **)&dev_b, N * sizeof(int));
	hipMalloc((void **)&dev_c, N * sizeof(int));

	// allocate page-locked memory , used to stream
	hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for(int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	// loop over full data, in bite-size chunks
	for(int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		hipMemcpyAsync(dev_a, host_a+i,N*sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b+i,N*sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_c, host_c+i,N*sizeof(int), hipMemcpyHostToDevice, stream);

		kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

		hipMemcpyAsync(host_c+i, dev_c, N*sizeof(int), hipMemcpyDeviceToHost, stream);
	}


	hipStreamSynchronize(stream);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "total time : " << elapsedTime << " ms" << endl;

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	// destroy stream
	hipStreamDestroy(stream);


	return 0;
}
*/
