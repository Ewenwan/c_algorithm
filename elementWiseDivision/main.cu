#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <hipblas.h>

using namespace std;

// declare constant memory
__constant__ float TestFA[4096];
__constant__ float TestFB[4096];
// declare texture reference
texture<float> texInA;
texture<float> texInB;
//texture<float> texOut;

__global__ void elementDivision(float *A, float *B, float *C)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int offset = x + y * blockDim.x * gridDim.x;

	C[offset] = A[offset] / B[offset];
}
__global__ void DivisionConst(float *SUM)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int offset = x + y * blockDim.x * gridDim.x;

	SUM[offset] = TestFA[offset] / TestFB[offset];
}
__global__ void DivisionTex(float *SUM)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int offset = x + y * blockDim.x * gridDim.x;

	// fetch data from texture
	float a = tex1Dfetch(texInA, offset);
	float b = tex1Dfetch(texInB, offset);

	SUM[offset] =  a / b;
}

int main(void)
{
	hipError_t cudaState = hipSuccess;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *TestConstA = new float [4096]();
	float *TestConstB = new float [4096]();
	float *SUM = new float [4096]();

	float *L_d, *R_d, *SUM_d;
	cudaState = hipMalloc((void **)&L_d, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&R_d, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&SUM_d, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);

	for(int i = 0; i < 4096; ++i)
	{
		TestConstA[i] = 2;
		TestConstB[i] = 1;
	}

	dim3 block(16, 16);
	dim3 grid(4096/256);

	float elapsedTime = 0;

	// do division on Constant Memory
	// copy data from host to constant memory
	hipEventRecord(start, 0);
	cudaState = hipMemcpyToSymbol(HIP_SYMBOL(TestFA), TestConstA, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpyToSymbol(HIP_SYMBOL(TestFB), TestConstB, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);
	// do division on constant memory
	DivisionConst<<<grid, block>>>(SUM_d);
	// copy data from device back to host
	cudaState = hipMemcpy(SUM, SUM_d, 4096 * sizeof(float), hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Constant Time : " << elapsedTime << " ms" << endl;

	for(int i = 0; i < 4096; ++i)
		if(SUM[i] == 0)
			cout << "There exist errors..." << endl;

	// do division on Global Memory
	// copy data from host to global memory
	hipEventRecord(start, 0);
	cudaState = hipMemcpy(L_d, TestConstA, 4096 * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(R_d, TestConstB, 4096 * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	// do division on GPU
	elementDivision<<<grid, block>>>(L_d, R_d, SUM_d);
	// copy data from device back to host
	cudaState = hipMemcpy(SUM, SUM_d, 4096 * sizeof(float), hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "global Time : " << elapsedTime << " ms" << endl;

	for(int i = 0; i < 4096; ++i)
		if(SUM[i] == 0)
			cout << "There exist errors..." << endl;

	hipEvent_t startT, stopT;
	hipEventCreate(&startT);
	hipEventCreate(&stopT);

	// do division on texture meory
	// copy data from host to global memory
	hipEventRecord(start, 0);
	cudaState = hipMemcpy(L_d, TestConstA, 4096 * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	cudaState = hipMemcpy(R_d, TestConstB, 4096 * sizeof(float), hipMemcpyHostToDevice);
	assert(cudaState == hipSuccess);
	// bind texture to global memory
	hipEventRecord(startT, 0);
	cudaState = hipBindTexture(NULL, texInA, L_d, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);
	cudaState = hipBindTexture(NULL, texInB, R_d, 4096 * sizeof(float));
	assert(cudaState == hipSuccess);
	hipEventRecord(stopT, 0);
	hipEventSynchronize(stopT);
	hipEventElapsedTime(&elapsedTime, startT, stopT);
	cout << "Bind texture memory:" << elapsedTime << "ms" << endl;
	// call the kernel
	DivisionTex<<<grid, block>>>(SUM_d);
	// copy result from texture back to hosts
	cudaState = hipMemcpy(SUM, SUM_d, sizeof(float) * 4096, hipMemcpyDeviceToHost);
	assert(cudaState == hipSuccess);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "texture Time : " << elapsedTime << " ms" << endl;

	for(int i = 0; i < 4096; ++i)
		if(SUM[i] == 0)
			cout << "There exist errors..." << endl;


	delete [] TestConstA;
	delete [] TestConstB;
	delete [] SUM;

	// Unbind the texture
	hipUnbindTexture(texInA);
	hipUnbindTexture(texInB);

	hipFree(R_d);
	hipFree(L_d);
	hipFree(SUM_d);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
